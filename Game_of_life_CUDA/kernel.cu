#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#define GLUT_DISABLE_ATEXIT_HACK
#include <cassert>
#include <iostream>
#include <fstream>
#include <Windows.h>
#include <stdio.h>
#include <helper_gl.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <gl/GL.h>
#include <gl/GLU.h>
#include <GL/freeglut.h>
#include <GLFW/glfw3.h>
#include "GUI.h"

constexpr u_int max_num_blocks = 32768;

using u_int = unsigned int;
using u_short = unsigned short;

u_int delay = 250;


__global__ void next_step(bool* map, bool* next_state, u_int width, u_int length, int32_t* image ) {
    u_int total_size = width * length;

    for (u_int idx = (u_int)(blockIdx.x * blockDim.x + threadIdx.x); idx < total_size; idx += (u_int)(blockDim.x * gridDim.x)) {
        // now we need to organize 2d in 1d array
        // for each element in map array we want to calculate number of alive neighbours
        // so we need to figure out how to indexing through given array
        // second problem is that we need to compute this but with respect of "curved borders" of the map

        // x positions of actual idx and right and left neighbour
        u_int x = idx % width;
        u_int x_left = (x + width - 1) % width;
        u_int x_right = (x + 1) % width;

        // y coords analogically as x
        u_int y = idx - x;
        u_int y_top = (y + total_size - width) % total_size;
        u_int y_bottom = (y + width) % total_size;

        u_short num_neigbhbour_alive = 0;

        // top neighbours
        num_neigbhbour_alive += (u_short)map[x_left + y_top] + (u_short)map[x + y_top] + (u_short)map[x_right + y_top];
        // left & right neighbours
        num_neigbhbour_alive += (u_short)map[x_left + y] + (u_short)map[x_right + y];
        // bottom neighbours
        num_neigbhbour_alive += (u_short)map[x_left + y_bottom] + (u_short)map[x + y_bottom] + (u_short)map[x_right + y_bottom];

        // compute state and set color
        next_state[x + y] = num_neigbhbour_alive == 3 || (num_neigbhbour_alive == 2 && map[x + y]);
        image[x + y] = next_state[x + y] ? -1 : 0;
    }
}


__host__ void calculate_map(bool*& map, bool*& next_state, u_int image_width, u_int image_height, u_short num_threads, int32_t*&out_data) {
    assert(image_width * image_height % num_threads == 0);
    u_int requested_blocks = ((image_width * image_height) / num_threads);
    u_int num_blocks = (u_int)min(max_num_blocks, requested_blocks);
    next_step <<< num_blocks, num_threads >>> (map, next_state, image_width, image_height, out_data);
    std::swap(map, next_state);
}



// copy image and process using CUDA
void generateCUDAImage(u_int image_width, u_int image_height, bool*& map, bool*& next_state, u_int num_threads, hipGraphicsResource* cuda_dest_resource, 
    hipGraphicsResource* cuda_tex_result_resource){
    // run the Cuda kernel
    int32_t* out_data;

    out_data = reinterpret_cast<int32_t*>(cuda_dest_resource);
    // calculate grid size
    dim3 block(16, 16, 1);
    dim3 grid(image_width / block.x, image_height / block.y, 1);
    calculate_map(map, next_state, image_width, image_height, num_threads, out_data);

    hipArray* texture_ptr;
    hipGraphicsMapResources(1, &cuda_tex_result_resource, 0);
    hipGraphicsSubResourceGetMappedArray(&texture_ptr, cuda_tex_result_resource, 0, 0);

    int num_texels = image_width * image_height;
    int num_values = num_texels * 4;
    int size_tex_data = sizeof(GLubyte) * num_values;
    hipMemcpyToArray(texture_ptr, 0, 0, cuda_dest_resource, size_tex_data, hipMemcpyDeviceToDevice);

    hipGraphicsUnmapResources(1, &cuda_tex_result_resource, 0);
}



void
display(u_int image_width, u_int image_height, bool*& map, bool*& next_state, u_int num_threads, GLint shDrawTex, GLuint tex_cudaResult, 
    hipGraphicsResource* cuda_dest_resource, hipGraphicsResource* cuda_tex_result_resource){
    generateCUDAImage(image_width, image_height, map, next_state, num_threads, cuda_dest_resource, cuda_tex_result_resource);
    displayImage(tex_cudaResult, shDrawTex);
    hipDeviceSynchronize();
}


void inputer(u_int& num_threads, u_int& image_width, u_int& image_height, u_int& total_size, bool*& host_map, bool*& host_next_state) {
    std::cout << "Welcome to the game of life!" << std::endl;
    std::cout << "Please choose f if you want to load data from file or press m to input data manually." << std::endl;
input:
    char mode = getchar();
    if (mode != 'f' && mode != 'm') {
        std::cout << "You choose wrong mode: " << mode << ". Please use f for file or m for manually." << std::endl;
        goto input;
    }
    else if (mode == 'f') {
        std::cout << "Now put the name of the file with size and start positions" << std::endl;
        std::string file_name;
        std::cin >> file_name;

        std::ifstream infile(file_name.c_str());
        if (!infile)
        {
            std::cout << "Cannot open file correctly" << std::endl;
        }
        if (!(infile >> num_threads)) {
            std::cout << num_threads << std::endl;
            std::cout << "Wrong file format, file must include number of threads" << std::endl;
            exit(1);
        }
        if (!(infile >> image_width >> image_height)) {
            std::cout << "Wrong file format, file must include width and height" << std::endl;
            exit(1);
        }
        total_size = image_width * image_height;
        host_map = new bool[total_size];
        host_next_state = new bool[total_size];
        for (u_int i = 0; i < total_size; ++i)
            host_map[i] = host_next_state[i] = false;
        u_int num_start_alive;
        if (!(infile >> num_start_alive)) {
            std::cout << "Wrong file format, file must include number of alive cells at the beginning" << std::endl;
            exit(1);
        }
        u_int x_pos, y_pos;
        u_int num_loaded = 0;
        while (infile >> x_pos >> y_pos) {
            if (x_pos >= image_width)
                x_pos = x_pos % image_width;
            if (y_pos >= image_height)
                y_pos = y_pos % image_height;
            num_loaded += 2;
            host_map[x_pos * image_width + y_pos] = true;
        }
        if (num_loaded < 2 * num_start_alive) {
            std::cout << "File does not contains enough coords" << std::endl;
            delete[] host_map;
            delete[] host_next_state;
            exit(1);
        }
        infile.close();
    }
    else {
        std::cout << "Please input number of threads" << std::endl;
        std::cin >> num_threads;
        std::cout << "Now, please input width and length" << std::endl;
        std::cin >> image_width >> image_height;
        total_size = image_width * image_height;
        host_map = new bool[total_size];
        host_next_state = new bool[total_size];
        for (u_int i = 0; i < total_size; ++i)
            host_map[i] = host_next_state[i] = false;
        u_int counter = 0;
        u_int x_pos, y_pos;
        u_int num_start_alive;
        std::cout << "Input number of alive cells at the beginning" << std::endl;
        std::cin >> num_start_alive;
        std::cout << "Input starting data" << std::endl;
        while (counter != num_start_alive) {
            std::cin >> x_pos >> y_pos;
            if (x_pos >= image_width)
                x_pos = x_pos % image_width;
            if (y_pos >= image_height)
                y_pos = y_pos % image_height;
            counter++;
            host_map[x_pos * image_width + y_pos] = true;
        }
    }
}

void error_callback(int error, const char* description){
    fprintf(stderr, "Error: %s\n", description);
}


void speed_up() {
    delay = (int)delay * 0.9;
}

void slow_down() {
    delay = (int)delay * 1 / (0.9);
}

void default_delay() {
    delay = 250;
}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods){
    if (key == GLFW_KEY_F && action == GLFW_PRESS)
        speed_up();
    else if (key == GLFW_KEY_S && action == GLFW_PRESS)
        slow_down();
    else if (key == GLFW_KEY_D && action == GLFW_PRESS)
        default_delay();
}


int main(int argc, char** argv) {
    u_int image_width, image_height, total_size, num_threads;
    bool *map, *next_state;
    bool *host_map, *host_next_state;
    bool* result;
    struct hipGraphicsResource* cuda_dest_resource;
    struct hipGraphicsResource* cuda_tex_result_resource = NULL;
    inputer(num_threads, image_width, image_height, total_size, host_map, host_next_state);

    hipMalloc((void**)&map, sizeof(bool) * total_size);
    hipMalloc((void**)&next_state, sizeof(bool) * total_size);
    hipMalloc((void**)&cuda_dest_resource, sizeof(int32_t) * total_size);
    hipMemset(map, false, total_size);
    hipMemset(next_state, false, total_size);
    std::cout << "Do you want to start [Y/n]?" << std::endl;
    char decision;
    std::cin >> decision;
    if (decision != 'y' && decision != 'Y') {
        std::cout << "Ending" << std::endl;
        return 0;
    }

    glfwSetErrorCallback(error_callback);
    glfwInit();
    GLFWwindow* window = glfwCreateWindow(512, 512, "Okienko", NULL, NULL);
    //GLFWwindow* window = glfwCreateWindow(800, 800, "Okienko", NULL, NULL);
    glfwMakeContextCurrent(window);
    glClearColor(0.5, 0.5, 0.5, 1.0);
    glDisable(GL_DEPTH_TEST);

    GLuint tex_cudaResult;

    hipMemcpy(map, host_map, sizeof(bool) * total_size, hipMemcpyHostToDevice);
    hipMemcpy(next_state, host_next_state, sizeof(bool) * total_size, hipMemcpyHostToDevice);
    createTextureDst(&tex_cudaResult, image_width, image_height, cuda_tex_result_resource);

    glewInit();
    GLint shDraw = compileGLSLprogram(NULL, glsl_draw_fragshader_src);
    GLint shDrawTex = compileGLSLprogram(glsl_drawtex_vertshader_src, glsl_drawtex_fragshader_src);
    while (!glfwWindowShouldClose(window)) {
        display(image_width, image_height, map, next_state, num_threads, shDrawTex, tex_cudaResult, cuda_dest_resource, cuda_tex_result_resource);
        glfwSwapBuffers(window);
        glfwPollEvents();
        glfwSetKeyCallback(window, key_callback);
        Sleep(delay);
    }
    glfwTerminate();
    hipFree(map);
    hipFree(next_state);
    hipFree(cuda_dest_resource);
    return 0;
}

